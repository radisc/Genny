#include "hip/hip_runtime.h"
#include <thrust/sort.h>
#include <thrust/transform.h>
#include <thrust/host_vector.h>
#include <thrust/device_ptr.h>
#include <thrust/device_vector.h>
#include <thrust/device_allocator.h>
#include <thrust/functional.h>
#include <thrust/execution_policy.h>
#include <algorithm>
#include <cstdlib>
#include <float.h>
#include <ufl.h>

#include <hip/hip_runtime.h>
#include <cuda_device_runtime_api.h>
#include <hip/hip_runtime_api.h>


//struct YStar {
//	int index;
//	double lpValue;
//};
//
//struct Map {
//	int index;
//	double value;
//};
//
//struct Instance {
//    long n_facilities;
//    long n_clients;
//    double *fixed_costs;
//    double *costs; // c[i,j] = inst->cost[i*inst->n_facilities+j]
//};


//Used to check if there are any errors launching the kernel
void CUDAErrorCheck(){
	hipError_t error = hipGetLastError();
	if (error != hipSuccess){
		printf("CUDA error : %s (%d)\n", hipGetErrorString(error), error);
		exit(0);
	}
}

//inline int ypos(int j, const Instance *inst) {
//    return j;
//}
//

//__host__ __device__
//double cudaFunctionI(const double *xstar, int i, 	const thrust::device_vector< double >&	d_fixcs	,
//													const thrust::device_vector< double >& 	d_costs	,
//													const int	d_nfac	,
//													const int	d_ncli	){
//
//	double minLinkingCosts = DBL_MAX;
//
//	//for every facility
//	for(int j = 0 ; j < d_nfac; j++ ){
//
//		//cout << "xstar[ypos(" << j <<  ", &instance)]: " <<  xstar[ypos(j, &instance)] << endl;
//
//		//(Opened facility)
//		if( ( xstar[j] ) > 1e-5 ){
//
//			//cout << "M:(" << i+1 << "," << j+1 << "): " << instance.costs[i*instance.n_facilities + j] << endl;
//
//			if(d_costs[i*d_nfac + j] < minLinkingCosts)
//				minLinkingCosts = d_costs[i*d_nfac + j];
//				//cout << "found minimum: " << minLinkingCosts << endl;
//				//cout << "M:(" << i+1 << "," << j+1 << "): " << instance.costs[i*instance.n_facilities + j] << endl;
//
//
//		}
//	}
//
//	return minLinkingCosts;
//}

//double cudaFunctionI(const double *xstar, int i,	thrust::device_vector< double >&	d_fixcs	,
//													thrust::device_vector< double >& 	d_costs	,
//													int			d_nfac	,
//													int			d_ncli	){
//
//	double minLinkingCosts = DBL_MAX;
//
//	//for every facility
//	for(int j = 0 ; j < d_nfac; j++ ){
//
//		//cout << "xstar[ypos(" << j <<  ", &instance)]: " <<  xstar[ypos(j, &instance)] << endl;
//
//		//(Opened facility)
//		if( ( xstar[j] ) > 1e-5 ){
//
//			//cout << "M:(" << i+1 << "," << j+1 << "): " << instance.costs[i*instance.n_facilities + j] << endl;
//
//			if(d_costs[i*d_nfac + j] < minLinkingCosts)
//				minLinkingCosts = d_costs[i*d_nfac + j];
//				//cout << "found minimum: " << minLinkingCosts << endl;
//				//cout << "M:(" << i+1 << "," << j+1 << "): " << instance.costs[i*instance.n_facilities + j] << endl;
//
//
//		}
//	}
//
//	return minLinkingCosts;
//}

//// square<T> computes the square of a number f(x) -> x*x
//struct calc_fitness
//{
//    const Instance inst;
//
//    calc_fitness(Instance _instance) : inst(_instance) {}
//
//	__host__ __device__
//        float operator()(const double xstar) const {
//
//    		double z_value = 0.0;
//
//    		double objVal;
//
//			for(int j = 0; j < inst.n_facilities ; j++){
//
//				if(xstar[ypos(j, &inst)] > 0.0 ){
//					objVal += inst.fixed_costs[j];
//				}
//			}
//
//			for(int i = 0 ; i < inst.n_clients ; i++){
//				objVal += cudaFunctionI(inst, &xstar, i);
//			}
//
//			return objVal;
//    	}
//};

/*
void loadInst(Instance* inst){

	//Init and fill with inst.costs
	d_costs = inst->costs;//(inst->costs, inst->costs + (inst->n_clients * inst->n_facilities));

	//Init with costs cardinality
	d_costs_trspnd = inst->costs;

	//Init and fill with fixed_costs
	d_fixed_costs = inst->fixed_costs;

	//Manually "traspose"
	for( int i = 0 ; i < inst->n_clients ; i++){
		for( int j = 0 ; j < inst->n_facilities ; j++ ){
			d_costs_trspnd[j*inst->n_clients + i] = inst->costs[i*inst->n_facilities + j];
		}
	}

	//d_fixed_costs = inst->fixed_costs;

}
*/

//void transpose(double* lin_matrix, double* lin_t_matrix , int n_cols, int n_rows ){
//
//	if(lin_t_matrix == NULL){
//		lin_t_matrix = (double*)malloc(sizeof(double)*n_cols*n_rows );
//	}
//
//	for( int i = 0 ; i < n_rows ; i++){
//		for( int j = 0 ; j < n_cols ; j++){
//			lin_t_matrix[j*n_rows + i] = lin_matrix[i*n_cols + j];
//		}
//	}
//
//}

void loadInstance(Instance& instance,	double& d_fixcs	,
										double& d_costs,
										double& d_t_costs,
										int		d_nfac	,
										int 	d_ncli	){


	hipMalloc((void**) &d_fixcs	, sizeof(double)*instance.n_facilities);
	CUDAErrorCheck();
	hipMalloc((void**)	&d_costs	, sizeof(double)*instance.n_facilities*instance.n_clients);
	CUDAErrorCheck();
	hipMalloc((void**) &d_t_costs	, sizeof(double)*instance.n_facilities*instance.n_clients);
	CUDAErrorCheck();


	hipMemcpy(	&d_fixcs, instance.fixed_costs, sizeof(double)*d_nfac, hipMemcpyHostToDevice);
	CUDAErrorCheck();
	hipMemcpy( &d_costs, instance.costs, d_nfac*d_ncli*sizeof(double), hipMemcpyHostToDevice );
	CUDAErrorCheck();

	hipDeviceSynchronize();
}

//class calc_fitness{
//
//	const thrust::device_vector< double >& 	d_fixcs;
//	const thrust::device_vector< double >& 	d_t_costs;
//	const int			d_nfac;
//	const int			d_ncli;
//
//public:
//	calc_fitness( 	thrust::device_vector< double >& fixcs_d, thrust::device_vector< double >& costs_t_d,
//					int		nfac_d		, int ncli_d ) :
//		d_fixcs (fixcs_d),
//		d_t_costs (costs_t_d),
//		d_nfac (nfac_d),
//		d_ncli (ncli_d) {}
//
//	__host__ __device__
//	double operator()(const double xstar) const {
//
//		double objVal = 0.0;
//
//		for(int j = 0; j < d_nfac ; j++){
//
//			if(xstar[j] > 0.0 ){
//				objVal += d_fixcs[j];
//			}
//		}
//
//		for(int i = 0 ; i < d_ncli ; i++){
//
//
//			//objVal += cudaFunctionI(xstar, i, d_fixcs, d_costs, d_nfac, d_ncli);
//		}
//
//		return objVal;
//
//	}
//};

//__global__
//void compute_fitness(double* fitness, double* xStar, int n_ind, double* d_fixc	,
//																double* d_costs,
//																int n_fac		,
//																int n_cli ){
//
//	__shared__ double xstar[3000];
//
//	int k = blockDim.x*blockIdx.x + threadIdx.x;
//	fitness[k] = 0.0;
//
//	if(k < n_ind){
//
//		double Wi;
//		double Cij;
//		for(int j = 0 ; j < n_fac ; j++){
//			//fac = xStar[k*n_fac + j];
//			if(xStar[k*n_fac + j] > 1e-5){
//				fitness[k] += d_fixc[j];//*xStar[k*n_fac + j];	//fitness[k] += d_fixc[j];
//			}
//			xstar[j] = xStar[k*n_fac + j];
//		}
//
//		for(int i = 0 ; i < n_cli ;  i++){
//
//			//Mini-fI
//			Wi = DBL_MAX;
//			for( int j = 0 ; j < n_fac ; j++){
//				if(xstar[j] > 1e-5){
//					Cij = d_costs[i*n_fac + j];
//					if(Cij  < Wi ){
//						Wi = Cij;
//					}
//				}
//			}
//
//			fitness[k] += Wi;
//		}
//	}
//}

//__global__
//void compute_fitness(double* fitness, double* xStar, int n_ind, double* d_fixc	,
//																double* d_costs,
//																int n_fac		,
//																int n_cli ){
//
//
//	int k = blockDim.x*blockIdx.x + threadIdx.x;
//	fitness[k] = 0.0;
//
//	if(k < n_ind){
//
//		double Wi;
//		double Cij;
//		for(int j = 0 ; j < n_fac ; j++){
//			//fac = xStar[k*n_fac + j];
//			if(xStar[k*n_fac + j] > 1e-5){
//				fitness[k] += d_fixc[j];//*xStar[k*n_fac + j];	//fitness[k] += d_fixc[j];
//			}
//		}
//
//		for(int i = 0 ; i < n_cli ;  i++){
//
//			//Mini-fI
//			Wi = DBL_MAX;
//			for( int j = 0 ; j < n_fac ; j++){
//				if(xStar[k*n_fac + j] > 1e-5){
//					Cij = d_costs[i*n_fac + j];
//					if(Cij  < Wi ){
//						Wi = Cij;
//					}
//				}
//			}
//
//			fitness[k] += Wi;
//		}
//	}
//}


__global__
void compute_fitness(double* fitness, double* xStar, int n_ind, double* d_fixc, double* d_costs, int n_fac , int n_cli ){

	double Wi;
	double Cij;
	int c = 0;
	int opened[128] = {0};

	int k = blockDim.x*blockIdx.x + threadIdx.x;


	if(k < n_ind){

		fitness[k] = 0.0;

		for(int j = 0 ; j < n_fac ; j++){
			if(xStar[k*n_fac + j] > 1e-5){
				opened[c] = j;
				c++;
				fitness[k] += d_fixc[j];
			}
		}

		for(int i = 0 ; i < n_cli ;  i++){

			//Mini-fI
			c = 0;
			Wi = DBL_MAX;
			for( int j = 0 ; j < n_fac ; j++){
				//if(xStar[k*n_fac + j] > 1e-5){
				if( opened[c] == j ){
					Cij = d_costs[i*n_fac + j];
					if(Cij  < Wi ){
						Wi = Cij;
					}
					c++;
				}
			}
			fitness[k] += Wi;
		}
	}
}

//////////////////////////////////NOT WORKING////////////////////////////////////////
//__global__
//void compute_fitness(double* fitness, double* xStar, int n_ind, double* d_fixc	,
//																double* d_costs,
//																int n_fac		,
//																int n_cli ){
//
//	__shared__ double fitnesses[32];
//
//	int k = blockDim.x*blockIdx.x + threadIdx.x;
//	fitness[k] = 0.0;
//
//	if(k < n_ind){
//
//		double Wi;
//		double Cij;
//
//		for(int j = 0 ; j < n_fac ; j++){
//
//			if(xStar[k*n_fac + j] > 1e-5){
//				//int fi = d_fixc[j];
//				fitnesses[threadIdx.x] += d_fixc[j]*xStar[k*n_fac + j];	//fitness[k] += d_fixc[j];
//			}
//		}
//
//		__syncthreads();
//
//		for(int i = 0 ; i < n_cli ;  i++){
//
//			//Mini-fI
//			Wi = DBL_MAX;
//			for( int j = 0 ; j < n_fac ; j++){
//				if(xStar[k*n_fac + j] > 1e-5){
//					Cij = d_costs[i*n_fac + j];//*xStar[k*n_fac + j];
//					if( Cij < Wi ){
//						Wi = Cij;
//					}
//				}
//				//__syncthreads();
//			}
//
//			__syncthreads();
//			fitnesses[threadIdx.x] += Wi;
//		}
//	}
//	__syncthreads();
//
//	if(k < n_ind){
//		fitness[k] = fitnesses[threadIdx.x];
//	}
//}

void execute_compute_fitness_good(double* fitness, double* xStars, int n_ind,	double* d_fixc, double* d_costs,
																			int n_fac,		int n_cli ,
																			int blocks, 	int threads){

	int tpb = 32;
	//cout <<"n_ind: " << n_ind << endl;
	//double* t_xStars;
//	compute_fitness <<< 16 , n_ind/16 >>> (fitness, xStars, n_ind, d_fixc, d_costs, n_fac, n_cli);

	compute_fitness <<< n_ind/tpb, tpb >>> (fitness, xStars, n_ind, d_fixc, d_costs, n_fac, n_cli);

	hipDeviceSynchronize();
}

void execute_compute_fitness(double* fitness, double* xStars, int n_ind,	double* d_fixc, double* d_costs,
																			int n_fac,		int n_cli ,
																			int blocks, 	int threads){

	int tpb = 128;
	int gridsize = floor( (double) n_ind/tpb ) + 1 ;

	if( VERBOSE ){
		cout << "gridsize: " << gridsize << endl;
		cout <<"n_ind: " << n_ind << endl;
	}
	//	compute_fitness <<< 16 , n_ind/16 >>> (fitness, xStars, n_ind, d_fixc, d_costs, n_fac, n_cli);

	compute_fitness <<< gridsize, tpb >>> (fitness, xStars, n_ind, d_fixc, d_costs, n_fac, n_cli);

	hipDeviceSynchronize();
}

void hybridFitness(vector< Solution >& pop, double* fitness, 	double* d_fixc,
																double* d_costs,
																int n_fac,
																int n_cli,
																double* fixed_costs,
																double* costs){


		double n_ind = pop.size();
		double* d_xstar;
		double* d_fitness;

		hipMalloc((void**)&d_xstar, (n_fac*n_ind) * sizeof(double));
		CUDAErrorCheck();

		hipMalloc((void**)&d_fitness, 	sizeof(double)*n_ind);
		CUDAErrorCheck();

		vector< Solution > specials;

		int n_blocks;

		//Memcpy every xStar
		for(int k = 0 ; k < n_ind ; k++){



			int offset = k*n_fac;
			double* ystar = (double*)malloc(sizeof(double)*n_fac);
			//hipHostMalloc((void**)&ystar, sizeof(double)*n_fac);
			memcpy(ystar, pop[k].xStar, sizeof(double)*n_fac);
			//cout << k << "): "; prettyPrintXStar(ystar, n_fac);
			//cout << endl;
			//hipMemcpy(ystar, pop[k].xStar, sizeof(double)*n_fac, hipMemcpyHostToHost);

			if(pop[k].non_zero_count < 128 && !( k < floor(n_ind/32)*32 ) )
				hipMemcpy(d_xstar + offset , ystar, sizeof(double)*n_fac, hipMemcpyHostToDevice );
			else
				specials.push_back(pop[k]);

			CUDAErrorCheck();

			free(ystar);
			hipDeviceSynchronize();
		}

		execute_compute_fitness(d_fitness, d_xstar, n_ind, d_fixc, d_costs, n_fac, n_cli, 1, 1);
		CUDAErrorCheck();

		//CPUfitness(specials, fixed_costs, costs);

		hipMemcpy(fitness, d_fitness, sizeof(double)*n_ind, hipMemcpyDeviceToHost);
		CUDAErrorCheck();

		//Write fitnesses on sons
		for(int k = 0 ; k < pop.size() ; k++){
			pop[k].z_opt = fitness[k];
			//prettyPrintXStar(pop[k].xStar, n_fac); cout << ": " << pop[k].z_opt << endl;
		}

		hipFree(d_xstar);
		hipFree(d_fitness);

}

void cudaFitness(vector< Solution >& pop, double* fitness, 	double* d_fixc,
															double* d_costs,
															int n_fac,
															int n_cli){

	int n_ind = pop.size();
	double* d_xstar;
	double* d_fitness;

	hipMalloc((void**)&d_xstar, (n_fac*n_ind) * sizeof(double));
	CUDAErrorCheck();

	hipMalloc((void**)&d_fitness, 	sizeof(double)*n_ind);
	CUDAErrorCheck();

	//vector< Solution > specials;

	//Memcpy every xStar
	for(int k = 0 ; k < n_ind ; k++){

		int offset = k*n_fac;
		double* ystar = (double*)malloc(sizeof(double)*n_fac);
		//hipHostMalloc((void**)&ystar, sizeof(double)*n_fac);
		memcpy(ystar, pop[k].xStar, sizeof(double)*n_fac);
		//cout << k << "): "; prettyPrintXStar(ystar, n_fac);
		//cout << endl;
		//hipMemcpy(ystar, pop[k].xStar, sizeof(double)*n_fac, hipMemcpyHostToHost);
		//if(pop[k].non_zero_count < 128)
		hipMemcpy(d_xstar + offset , ystar, sizeof(double)*n_fac, hipMemcpyHostToDevice );
		//else
		//	specials.push_back(pop[k]);

		CUDAErrorCheck();

		free(ystar);
		hipDeviceSynchronize();
	}

	execute_compute_fitness(d_fitness, d_xstar, n_ind, d_fixc, d_costs, n_fac, n_cli, 1, 1);
	CUDAErrorCheck();

	//CPUfitness(specials, ins)

	hipMemcpy(fitness, d_fitness, sizeof(double)*n_ind, hipMemcpyDeviceToHost);
	CUDAErrorCheck();

	//Write fitnesses on sons
	for(int k = 0 ; k < n_ind ; k++){
		pop[k].z_opt = fitness[k];
		//prettyPrintXStar(pop[k].xStar, n_fac); cout << ": " << pop[k].z_opt << endl;
	}

	hipFree(d_xstar);
	hipFree(d_fitness);

}


void cudaFitness2(vector< Solution >& pop, double* fitness, 	double* d_fixc,
																double* d_costs,
																int n_fac,
																int n_cli){
	int n_ind = pop.size();
//	cout << "sons to calc: " << n_ind << endl;
//	if( true ){	//Split if too big ( or else Cuda Kernel goes timeout )
//	double* fitness = (double*)calloc(n_ind, sizeof(double));

	cudaFitness(pop, fitness, d_fixc, d_costs, n_fac, n_cli);//, d_fixc, d_costs, instance.n_facilities, instance.n_clients);

//			for(int k = 0 ; k < pop.size() ; k++ ){
//
//				cout << k << ")" << "GPU fitness: " << fitnesses[k] << " CPU fitness: " << pop[k].z_opt << endl;
//
//			}
	//fitness = fitnesses;

	//free(fitnesses);

//	} else {

//		vector< Solution > sons_c;
//
//		int CHUNCK_SIZE = 128;
//		//int N_CHUNCKS =	n_ind/CHUNCK_SIZE;
//
//		//Create Chuncks
//		for(int c = 0 ; c < ceil(n_ind/CHUNCK_SIZE) ; c++ ){
//
//			vector< Solution > chunck (CHUNCK_SIZE);
//			double* fitnesses = (double*)malloc(sizeof(double)*CHUNCK_SIZE);
//			copy(pop.begin() + CHUNCK_SIZE*c, pop.begin() + CHUNCK_SIZE*(c+1), chunck.begin() );
//			cudaFitness(chunck, fitnesses, d_fixc, d_costs, n_fac, n_cli);
//			free(fitnesses);
//			sons_c.insert(sons_c.end(), chunck.begin(), chunck.end());
//
//		}
//		pop = sons_c;

//	}

	//cout << "after: " << pop.size();

}



//void cudaSort(Instance* inst, double* ystar, YStar** y_ord){
//
//	//Initializing and loading device vectors
//	thrust::device_vector<int> 		d_indexes( inst->n_facilities);//	((int) inst->n_facilities);
//	thrust::device_vector<double> 	d_lpValues(ystar, ystar + inst->n_facilities) ;//	((int) inst->n_facilities);
//
////	thrust::copy(ystar, ystar + ( (uint)inst->n_facilities*sizeof(double) ), d_lpValues.begin());
//
//	//Filling index vector from 0 to (#Facilities-1)
//	thrust::sequence(d_indexes.begin(), d_indexes.end());
//
///*
//	for (int k = 0; k < inst->n_facilities; k++) {
//		d_lpValues[k] = ystar[k];
//	}
//
//	//thrust::copy(ystar, ystar + inst->n_facilities, d_lpValues);
//	//hipMemcpy(&d_lpValues, ystar, sizeof(double) * inst->n_facilities, hipMemcpyHostToDevice);
//
//	//d_lpValues = ystar;
//
//	//printf("sfkjghfelkgjfhgkjf\n");
//*/
//
//	//SORT!
//	thrust::stable_sort_by_key(d_lpValues.begin(), d_lpValues.end() , d_indexes.begin(), thrust::greater<double>());
//
//	//Device2Host copy
//	thrust::host_vector<int> index = d_indexes;
//	thrust::host_vector<double> lp_Values = d_lpValues;
//
//	//Re-Create array of structures
//	for (int k = 0; k < inst->n_facilities; k++) {
//		y_ord[k] = (YStar *) malloc(sizeof(YStar));
//		y_ord[k]->index = index[k];
//		y_ord[k]->lpValue = lp_Values[k];
//	}
//}

//void cudaFitness(Instance& inst, vector< Solution >& population, double* fitness){
//
//	thrust::device_vector<double*> 	d_xstars(population.size());
//	thrust::device_vector<double >	d_f_values(population.size());
//
//	//Upload xstars
//	for (int k = 0; k < population.size(); k++) {
//		d_xstars[k] = population[k].xStar;
//	}
//	thrust::transform(d_xstars.begin(), d_xstars.end(), d_f_values.begin() , calc_fitness<double*>(d_xstars, d_f_values ));
//
//}

//void cudaSort(Instance* inst, double* ystar, Map** y_ord){
//
//	//YStar **y_ord = (YStar **) malloc(inst->n_facilities * sizeof(YStar*));
//
//	//Initializing and loading device vectors
//	thrust::device_vector<int> 		d_indexes(inst->n_facilities);//	((int) inst->n_facilities);
//	thrust::device_vector<double> 	d_lpValues(ystar, ystar + inst->n_facilities) ;//	((int) inst->n_facilities);
//
//	//Filling index vector from 0 to (#Facilities-1)
//	thrust::sequence(d_indexes.begin(), d_indexes.end());
//
//	//SORT!
//	thrust::stable_sort_by_key(d_lpValues.begin(), d_lpValues.end() , d_indexes.begin(), thrust::greater<double>());
//
//	//Device2Host copy
//	thrust::host_vector<int> index = d_indexes;
//	thrust::host_vector<double> lp_Values = d_lpValues;
//
//	//Re-Create array of structures
//	for (int k = 0; k < inst->n_facilities; k++) {
//		y_ord[k] = (Map *) malloc(sizeof(Map));
//		y_ord[k]->index = index[k];
//		y_ord[k]->value = lp_Values[k];
//	}
//}
